#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include "main.h"
#include "update.h"
#include "SIPLmini/SIPLmini.cpp"

//to calculate runtime
//#include <cstdio> 
//#include <ctime> 
//clock_t start;
//double duration;

using namespace std;
//using namespace SIPL;

float image[HEIGHT][WIDTH][DEPTH] = { 0 }; //input -> image to be segmented
float phi[HEIGHT][WIDTH][DEPTH] = { 0 };
int init[HEIGHT][WIDTH][DEPTH] = { 0 };
int label[HEIGHT][WIDTH][DEPTH] = { 0 };
int zeroLevelSet[HEIGHT][WIDTH][DEPTH] = { 0 }; //output
int layer[HEIGHT][WIDTH][DEPTH]; //-> see main.h for details

int iterations;
float threshold, alpha, epsilon;

float *phiD;
int *labelD;
int *layerD;
float *imageD;

__device__ float thresholdD, alphaD, epsilonD;

#define cudaCheckErrors(msg) \
    do { \
        hipError_t __err = hipGetLastError(); \
        if (__err != hipSuccess) { \
            fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n", \
                msg, hipGetErrorString(__err), \
                __FILE__, __LINE__); \
            fprintf(stderr, "*** FAILED - ABORTING\n"); \
            exit(1); \
        } \
    } while (0)

//fills init with the seed points, returns 1 if success
void fillSphere(SIPL::int3 seed, int radius){
	for(int i = seed.x - radius; i<seed.x + radius; i++){
	for(int j = seed.y - radius; j<seed.y + radius; j++){
	for(int k = seed.z - radius; k<seed.z + radius; k++){
		SIPL::int3 n(i,j,k);
		if(sqrt((float)((seed.x-n.x)*(seed.x-n.x)+(seed.y-n.y)*(seed.y-n.y)+(seed.z-n.z)*(seed.z-n.z))) < radius){
			init[i][j][k] = 1;
		}
	}}}
}

/* returns true if any neighbour of coordinates (i,j,j) in either
   init[][][] (id = 1) or label[][][] (id = 2) equals res */
bool checkMaskNeighbours(int i, int j, int k, short res){
	if(init[i+1][j][k] == res)
		return true;
	else if(init[i-1][j][k] == res) 
		return true;
	else if(init[i][j+1][k] == res)
		return true;
	else if(init[i][j-1][k] == res) 
		return true;
	else if(init[i][j][k+1] == res) 
		return true;
	else if(init[i][j][k-1] == res)
		return true;
	return false;
}

//add pixels to lists according to their label
void assignLabel(int i, int j, int k, int level){
	switch(level){
	case 1:
		layer[i][j][k] = 16; //add to lp1
		label[i][j][k] = level;
		phi[i][j][k] = level;
		break;
	case 2:
		layer[i][j][k] = 17; //add to lp2
		label[i][j][k] = level;
		phi[i][j][k] = level;
		break;
	case -1:
		layer[i][j][k] = 14; //add to ln1
		label[i][j][k] = level;
		phi[i][j][k] = level;
		break;
	case -2:
		layer[i][j][k] = 13; //add to ln2
		label[i][j][k] = level;
		phi[i][j][k] = level;	
		break;
	}
}

void setLevels(int i, int j, int k, int level){
	if(label[i+1][j][k] == 3){
		assignLabel(i+1, j, k, level);
	}
	if(label[i][j+1][k] == 3){
		assignLabel(i, j+1, k, level);
	}
	if(label[i-1][j][k] == 3){
		assignLabel(i-1, j, k, level);
	}
	if(label[i][j-1][k] == 3){
		assignLabel(i, j-1, k, level);
	}
	if(label[i][j][k+1] == 3){
		assignLabel(i, j, k+1, level);
	}
	if(label[i][j][k-1] == 3){
		assignLabel(i, j, k-1, level);
	}
	
	if(label[i+1][j][k] == -3){
		assignLabel(i+1, j, k, -level);
	}
	if(label[i][j+1][k] == -3){
		assignLabel(i, j+1, k, -level);
	}
	if(label[i-1][j][k] == -3){
		assignLabel(i-1, j, k, -level);
	}
	if(label[i][j-1][k] == -3){
		assignLabel(i, j-1, k, -level);
	}
	if(label[i][j][k+1] == -3){
		assignLabel(i, j, k+1, -level);
	}
	if(label[i][j][k-1] == -3){
		assignLabel(i, j, k-1, -level);
	}
}		

//initializes Ln2, Ln1, Lz, Lp1, Lp2 based on seed point(s) in init[][][]
void initialization(){
	for (int i = 0; i<HEIGHT; i++){
		for (int j = 0; j<WIDTH; j++){
			for (int k = 0; k<DEPTH; k++){
				if(init[i][j][k] == 0){
					label[i][j][k] = 3; 
					phi[i][j][k] = 3;
				}
				else{
					label[i][j][k] = -3; 
					phi[i][j][k] = -3;
				}
			}
		}
	}
	for (int i = 1; i<HEIGHT+1; i++){
		for (int j = 1; j<WIDTH+1; j++){
			for (int k = 0; k<DEPTH+1; k++){
				if(init[i][j][k] == 1 && checkMaskNeighbours(i, j, k, 0) == true){
					layer[i][j][k] = 15; //add to lz
					label[i][j][k] = 0;
					phi[i][j][k]= 0;
				}
			}
		}
	}
	for (int i = 0; i<HEIGHT; i++){
		for (int j = 0; j<WIDTH; j++){
			for (int k = 0; k<DEPTH; k++){
				if(layer[i][j][k] == 15){ //lz
					setLevels(i, j, k, 1);
				}
			}
		}
	}
	for (int i = 0; i<HEIGHT; i++){
		for (int j = 0; j<WIDTH; j++){
			for (int k = 0; k<DEPTH; k++){
				if(layer[i][j][k] == 16){// lp1
					setLevels(i, j, k, 2);
				}
			}
		}
	}
	for (int i = 0; i<HEIGHT; i++){
		for (int j = 0; j<WIDTH; j++){
			for (int k = 0; k<DEPTH; k++){
				if(layer[i][j][k] == 14){ //ln1
					setLevels(i, j, k, 2);
				}
			}
		}
	}
}

//allocate and copy data to device
void setUpDeviceArrays(){
	const size_t arrSize = size_t(HEIGHT*WIDTH*DEPTH);
	hipMalloc((void**)&phiD, sizeof(float)*arrSize);
	cudaCheckErrors(" hipMalloc error1");
	hipMalloc((void**)&labelD, sizeof(int)*arrSize);
	cudaCheckErrors(" hipMalloc error2");
	hipMalloc((void**)&layerD, sizeof(int)*arrSize);
	cudaCheckErrors(" hipMalloc error3");
	hipMalloc((void**)&imageD, sizeof(float)*arrSize);
	cudaCheckErrors(" hipMalloc error4");
	
	hipMemcpy(phiD, phi, sizeof(float)*arrSize, hipMemcpyHostToDevice);
	cudaCheckErrors(" hipMemcpy error1");
	hipMemcpy(labelD, label, sizeof(int)*arrSize, hipMemcpyHostToDevice);
	cudaCheckErrors(" hipMemcpy error2");
	hipMemcpy(layerD, layer, sizeof(int)*arrSize, hipMemcpyHostToDevice);
	cudaCheckErrors(" hipMemcpy error3");
	hipMemcpy(imageD, image, sizeof(float)*arrSize, hipMemcpyHostToDevice);
	cudaCheckErrors(" hipMemcpy error4");
	
}

	
void displayUshortVolume(SIPL::Volume<SIPL::ushort> * V){ 
	//Volume<float2> * v2 = new Volume<float2>(V->getSize());
	for(int x = 0; x < V->getWidth(); x++) {
	for(int y = 0; y < V->getHeight(); y++) {
	for(int z = 0; z < V->getDepth(); z++) {
		SIPL::int3 n(x,y,z);	
		image[x][y][z] = (int)(x+y*V->getWidth()+z*V->getWidth()*V->getHeight())/ 2000.0f;
		if(image[x][y][z] > 1.0f){
			image[x][y][z] = 1.0f;
		}
	}}}
	
}

void displayUcharVolume(SIPL::Volume<SIPL::uchar> * V){ 
	for(int x = 0; x < V->getWidth(); x++) {
	for(int y = 0; y < V->getHeight(); y++) {
	for(int z = 0; z < V->getDepth(); z++) {					
		image[x][y][z] = (float)((int)V->data[x+y*WIDTH+z*WIDTH*HEIGHT]/ 255.0f);
		
	}}}
}

bool getAndVerifyInput(int argc, char *argv[]){
	if(argc != 5){
		printf("Need four inputs: iterations, threshold, epsilon, alpha \n");
		return false;
	}
	if(sscanf (argv[1], "%i", & iterations)!=1 || iterations<0) {
		printf("Need four inputs: iterations, threshold, epsilon, alpha \n");
		return false;
	}
	if(sscanf(argv[2], "%f", &threshold)!=1 || threshold >1){
		printf("Need four inputs: iterations, threshold, epsilon, alpha \n");
		return false;
	}
	if(sscanf(argv[3], "%f", &epsilon)!=1 || epsilon >1){
		printf("Need four inputs: iterations, threshold, epsilon, alpha \n");
		return false;
	}
	if(sscanf(argv[4], "%f", &alpha)!=1 || alpha>1){
		printf("Need four inputs: iterations, threshold, epsilon, alpha \n");
		return false;
	}
	return true;
}

__global__ void lol(int *labelD, int a){
	int kk = 0;
	for(int i = 0; i<HEIGHT; i++){
		for(int j = 0; j < WIDTH; j++){
			for(int k = 0; k<DEPTH; k++){
				if(labelD[i + WIDTH * (j + DEPTH * k)] == 0){
					kk++;
				}
	}}}
	if(a == 1){
		printf("before labelD: %i\n", kk);
	}
	if(a == 2){
		printf("after labelD: %i\n", kk);
	}
}

int main(int argc, char *argv[]){
	if(!getAndVerifyInput(argc, argv)){
		system("pause");
		return 0;
	}
	
	
	//SIPL::int3 seed(139, 69, 78);
	//SIPL::int3 seed2(173, 89, 127);
	
	//SIPL::int3 seed(110, 107, 162);
	SIPL::int3 seed(103, 118, 128);
	fillSphere(seed, 10);
	//fillSphere(seed2, 3);
	
	//SIPL::Volume<SIPL::ushort> * V = new SIPL::Volume<SIPL::ushort>("t1_kontrast.raw", 256,256,192);
	SIPL::Volume<SIPL::uchar> * V = new SIPL::Volume<SIPL::uchar>("circle_with_values_245.raw", 256, 256, 256);
	
	initialization();
	setUpDeviceArrays(); //copy over arrays to device
	setVariablesInDevice<<<1,1>>>(threshold, epsilon, alpha);
	
	//displayUshortVolume(V);
	displayUcharVolume(V);

	const dim3 BlockDim(16, 4, 4);
    dim3 GridDim;
    GridDim.x = (WIDTH + BlockDim.x - 1) / BlockDim.x;
    GridDim.y = (HEIGHT + BlockDim.y - 1) / BlockDim.y;
	GridDim.z = (DEPTH + BlockDim.z - 1) / BlockDim.z;
	hipDeviceSynchronize();
	int cc = 0;
	for(int i = 0; i<HEIGHT; i++){
		for(int j = 0; j < WIDTH; j++){
			for(int k = 0; k<DEPTH; k++){
				if(label[i][j][k] == 0){
					cc++;
				}
	}}}
	lol<<<1,1>>>(labelD, 1);
	printf("before label %i\n", cc);
	hipDeviceSynchronize();
	
	printf("starting main loop\n");
	//start = std::clock();
	for(int i=1; i<iterations+1; i++){
		if(i%100 == 0){
			printf("iteration: %i\n", i);
		}
		prepareUpdates1<<<GridDim, BlockDim>>>(phiD, layerD, imageD);
		prepareUpdates2<<<GridDim, BlockDim>>>(phiD, layerD);
		prepareUpdates3<<<GridDim, BlockDim>>>(phiD, layerD, labelD);
		prepareUpdates4<<<GridDim, BlockDim>>>(phiD, layerD, labelD);
		prepareUpdates5<<<GridDim, BlockDim>>>(phiD, layerD, labelD);
		updateLevelSets1<<<GridDim, BlockDim>>>(phiD, layerD, labelD);
		updateLevelSets2<<<GridDim, BlockDim>>>(layerD, labelD);
	}
	hipDeviceSynchronize();
	
	//duration = ( std::clock() - start ) / (double) CLOCKS_PER_SEC;
	printf("\nmain loop finished\n");
	//printf("time used: %f\n", duration);
	//copy back result from device
	hipMemcpy(label, labelD, sizeof(int)*(size_t)(HEIGHT*WIDTH*DEPTH), hipMemcpyDeviceToHost);
	cudaCheckErrors("hipMemcpyDeviceToHost error");

	hipDeviceSynchronize();
	
	
	lol<<<1,1>>>(labelD, 2);
	hipDeviceSynchronize();
	int tt=0;
	for (int i = 0; i<HEIGHT; i++){
		for (int j = 0; j<WIDTH; j++){
			for (int k = 0; k<DEPTH; k++){
				if(label[i][j][k] == 0){ //lz
					zeroLevelSet[i][j][k] = 255;
					tt++;
				}
			}
		}
	}printf("after label: %i\n", tt);
	SIPL::Volume<SIPL::uchar> * v3 = new SIPL::Volume<SIPL::uchar>(V->getSize());
	for(int x = 0; x < V->getWidth(); x++) {
	for(int y = 0; y < V->getHeight(); y++) {
	for(int z = 0; z < V->getDepth(); z++) {
	SIPL::int3 n(x,y,z);	
		v3->set(n, (SIPL::uchar)zeroLevelSet[x][y][z]);
	}}}

	v3->save("resultCUDA.raw");
	printf("file stored\n");
	hipFree(phiD);
	hipFree(labelD);
	hipFree(layerD);
	hipFree(imageD);
	system("pause");
	return 0;
}
