#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include <cmath>
#include <hip/hip_runtime.h>
#include <exception>
#include <algorithm>
#include "main.h"
#include "update.h"
using namespace std;

extern float *phiD;
extern int *labelD;
extern int *layerD;
extern float *imageD;
extern __device__ float thresholdD, epsilonD, alphaD;

#define index(i,j,k) (i + WIDTH * (j + DEPTH * k))

//[ i*cols+j + rows*cols*z] ???
//[x * 256 * 256 + y * 256 + z] ???
//Flat[x + WIDTH * (y + DEPTH * z)] = Original[x, y, z] -> TEST UT DENNE F�RST

//nvcc --machine 32 -arch sm_20 dd.cpp main.cu update.cu 

void __global__ setVariablesInDevice(float threshold, float epsilon, float alpha){
	thresholdD = threshold;
	epsilonD = epsilon;
	alphaD = alpha;
}

//Returns either max or min (based on greaterOrLess) of the neighbours, with values less or greater than checkAgainst
__device__ float minMax(int i, int j, int k, int greaterOrLess, int checkAgainst, float *phiD, int *labelD){
	float minMaxRes = checkAgainst;
	if(greaterOrLess == 1){
		if(labelD[index(i+1,j,k)] >= minMaxRes){
			minMaxRes = phiD[index(i+1,j,k)];
		}
		if(labelD[index(i,j+1,k)] >= minMaxRes){
			minMaxRes = phiD[index(i,j+1,k)];
		}
		if(labelD[index(i-1,j,k)] >= minMaxRes){
			minMaxRes = phiD[index(i-1,j,k)];
		}
		if(labelD[index(i,j-1,k)] >= minMaxRes){
			minMaxRes = phiD[index(i,j-1,k)];
		}
		if(labelD[index(i,j,k+1)] >= minMaxRes){
			minMaxRes = phiD[index(i,j,k+1)];
		}
		if(labelD[index(i,j,k-1)] >= minMaxRes){
			minMaxRes = phiD[index(i,j,k-1)];
		}
	}
	else if(greaterOrLess == -1){
		if(labelD[index(i+1,j,k)] <= minMaxRes){
			minMaxRes = phiD[index(i+1,j,k)];
		}
		if(labelD[index(i,j+1,k)] <= minMaxRes){
			minMaxRes = phiD[index(i,j+1,k)];
		}
		if(labelD[index(i-1,j,k)] <= minMaxRes){
			minMaxRes = phiD[index(i-1,j,k)];
		}
		if(labelD[index(i,j-1,k)] <= minMaxRes){
			minMaxRes = phiD[index(i,j-1,k)];
		}
		if(labelD[index(i,j,k+1)] <= minMaxRes){
			minMaxRes = phiD[index(i,j,k+1)];
		}
		if(labelD[index(i,j,k-1)] <= minMaxRes){
			minMaxRes = phiD[index(i,j,k-1)];
		}
	}
	return minMaxRes;	
}

__device__ bool checkMaskNeighbours2(int i, int j, int k, int res, int *labelD){
	if(labelD[index(i+1,j,k)] == res)
		return true;
	else if(labelD[index(i-1,j,k)] == res)
		return true;
	else if(labelD[index(i,j+1,k)] == res)
		return true;
	else if(labelD[index(i,j-1,k)] == res)
		return true;
	else if(labelD[index(i,j,k+1)] == res)
		return true;
	else if(labelD[index(i,j,k-1)] == res)
		return true;
	return false;
	
}

__device__ float speedFunction(int i, int j, int k, float *phiD, float *imageD){ 
	//calculate data term
	float data = epsilonD - abs(imageD[index(i,j,k)] - thresholdD); //the data term (based on pixel intensity)
	//calculate 1-order derivatives
	float dx = (phiD[index(i+1,j,k)] - phiD[index(i-1,j,k)]) / 2;
	float dy = (phiD[index(i,j+1,k)] - phiD[index(i,j-1,k)]) / 2;
	float dz =  (phiD[index(i,j,k+1)] - phiD[index(i,j,k-1)]) / 2;
	float dxPlus  = phiD[index(i+1,j,k)] - phiD[index(i,j,k)];
	float dyPlus  = phiD[index(i,j+1,k)] - phiD[index(i,j,k)];
	float dzPlus  = phiD[index(i,j,k+1)] - phiD[index(i,j,k)];
	float dxMinus = phiD[index(i,j,k)] - phiD[index(i-1,j,k)];
	float dyMinus = phiD[index(i,j,k)] - phiD[index(i,j-1,k)];
	float dzMinus = phiD[index(i,j,k)] - phiD[index(i,j,k-1)];
	//calculate 2-order derivatives
	float dxPlusY  = (phiD[index(i+1,j+1,k)] - phiD[index(i-1,j+1,k)]) / 2;
	float dxMinusY = (phiD[index(i+1,j-1,k)] - phiD[index(i-1,j-1,k)]) / 2;
	float dxPlusZ  = (phiD[index(i+1,j,k+1)] - phiD[index(i-1,j,k+1)]) / 2;
	float dxMinusZ = (phiD[index(i+1,j,k-1)] - phiD[index(i-1,j,k-1)]) / 2;
	float dyPlusX  = (phiD[index(i+1,j+1,k)] - phiD[index(i+1,j-1,k)]) / 2;
	float dyMinusX = (phiD[index(i-1,j+1,k)] - phiD[index(i-1,j-1,k)]) / 2;
	float dyPlusZ  = (phiD[index(i,j+1,k+1)] - phiD[index(i,j-1,k+1)]) / 2;
	float dyMinusZ = (phiD[index(i,j+1,k-1)] - phiD[index(i,j-1,k-1)]) / 2;
	float dzPlusX  = (phiD[index(i+1,j,k+1)] - phiD[index(i+1,j,k-1)]) / 2;
	float dzMinusX = (phiD[index(i-1,j,k+1)] - phiD[index(i-1,j,k-1)]) / 2;
	float dzPlusY  = (phiD[index(i,j+1,k+1)] - phiD[index(i,j+1,k-1)]) / 2;
	float dzMinusY = (phiD[index(i,j-1,k+1)] - phiD[index(i,j-1,k-1)]) / 2;
	//calculate normals
	float nPlusX = dxPlus / sqrt(dxPlus * dxPlus + pow((dyPlusX + dy) / 2, 2) + pow((dzPlusX + dz) / 2, 2));
	float nPlusY = dyPlus / sqrt(dyPlus * dyPlus + pow((dxPlusY + dx) / 2, 2) + pow((dzPlusY + dz) / 2, 2));
	float nPlusZ = dzPlus / sqrt(dzPlus * dzPlus + pow((dxPlusZ + dx) / 2, 2) + pow((dyPlusZ + dy) / 2, 2));
	float nMinusX = dxMinus / sqrt(dxMinus * dxMinus + pow((dyMinusX + dy) / 2, 2) + pow((dzMinusX + dz) / 2, 2));
	float nMinusY = dyMinus / sqrt(dyMinus * dyMinus + pow((dxMinusY + dx) / 2, 2) + pow((dzMinusY + dz) / 2, 2));
	float nMinusZ = dzMinus / sqrt(dzMinus * dzMinus + pow((dxMinusZ + dx) / 2, 2) + pow((dyMinusZ + dy) / 2, 2));
	//calculate curvature
	float curvature = (nPlusX - nMinusX) + (nPlusY - nMinusY) + (nPlusZ - nMinusZ);
	//calculate the speeed
	float speed = -alphaD*data + (1-alphaD)*(curvature/8);
	if(speed > 1){
		speed = 1;
	}
	if(speed < -1){
		speed = -1;
	}
	return speed;
}


__global__ void prepareUpdates1(float *phiD, int *layerD, float *imageD){
	int i = threadIdx.x + blockDim.x * blockIdx.x;
	int j = threadIdx.y + blockDim.y * blockIdx.y;
	int k = threadIdx.z + blockDim.z * blockIdx.z;
	if(layerD[index(i,j,k)] == 15){ //lz
		phiD[index(i,j,k)] += speedFunction(i,j,k, phiD, imageD);
	}	
}

__global__ void prepareUpdates2(float *phiD, int *layerD){
	int i = threadIdx.x + blockDim.x * blockIdx.x;
	int j = threadIdx.y + blockDim.y * blockIdx.y;
	int k = threadIdx.z + blockDim.z * blockIdx.z;
	if(layerD[index(i,j,k)] == 15){ //lz
		if(phiD[index(i,j,k)] >= 0.5){
			layerD[index(i,j,k)] = 26; //add to sp1
		}
		else if(phiD[index(i,j,k)] < -0.5){
			layerD[index(i,j,k)] = 24; //add to sn1
		}
	}	
}

__global__ void prepareUpdates3(float *phiD, int *layerD, int *labelD){
	int i = threadIdx.x + blockDim.x * blockIdx.x;
	int j = threadIdx.y + blockDim.y * blockIdx.y;
	int k = threadIdx.z + blockDim.z * blockIdx.z;
	if(i !=0 && j !=0 && i < HEIGHT-1 && j < WIDTH-1){
		float M = 0;
		if(layerD[index(i,j,k)] == 14){ //ln1
			if(checkMaskNeighbours2(i, j, k, 0, labelD) == false){
				layerD[index(i,j,k)] = 23; //add to sn2
			}
			else{
				M = minMax(i, j, k,1, 0, phiD, labelD);
				phiD[index(i,j,k)] = M-1;
				if(phiD[index(i,j,k)] >= -0.5){
					layerD[index(i,j,k)] = 25; //add to sz
				}
				else if(phiD[index(i,j,k)] < -1.5){
					layerD[index(i,j,k)] = 23; //add to sn2
				}
			}
		}
	}
}

__global__ void prepareUpdates4(float *phiD, int *layerD, int *labelD){
	int i = threadIdx.x + blockDim.x * blockIdx.x;
	int j = threadIdx.y + blockDim.y * blockIdx.y;
	int k = threadIdx.z + blockDim.z * blockIdx.z;
	if(i !=0 && j !=0 && i < HEIGHT-1 && j < WIDTH-1){
		float M = 0;
		if(layerD[index(i,j,k)] == 16){ //lp1
			if(checkMaskNeighbours2(i, j, k, 0, labelD) == false){
				layerD[index(i,j,k)] = 27; //add to sp2
			}
			else{
				M = minMax(i, j, k,-1, 0, phiD, labelD);
				phiD[index(i,j,k)] = M+1;
				if(phiD[index(i,j,k)] < 0.5){
					layerD[index(i,j,k)] = 25; //add to sz
				}
				else if(phiD[index(i,j,k)] >= 1.5){
					layerD[index(i,j,k)] = 27; //add to sp2
				}
			}
		}
	}
}

__global__ void prepareUpdates5(float *phiD, int *layerD, int *labelD){
	int i = threadIdx.x + blockDim.x * blockIdx.x;
	int j = threadIdx.y + blockDim.y * blockIdx.y;
	int k = threadIdx.z + blockDim.z * blockIdx.z;
	if(i !=0 && j !=0 && i < HEIGHT-1 && j < WIDTH-1){
		float M = 0;
		if(layerD[index(i,j,k)] == 13){ //ln2
			if(checkMaskNeighbours2(i, j, k, -1, labelD) == false){
				labelD[index(i,j,k)] = -3;
				phiD[index(i,j,k)] = -3;
				layerD[index(i,j,k)] = 0; //no longer part of ln2
			}
			else{
				M = minMax(i, j, k, 1, -1, phiD, labelD);
				phiD[index(i,j,k)] = M-1;
				if(phiD[index(i,j,k)] >= -1.5){
					layerD[index(i,j,k)] = 24; //add to sn1
				}
				else if(phiD[index(i,j,k)] < -2.5){
					labelD[index(i,j,k)] = -3;
					phiD[index(i,j,k)] = -3;
					layerD[index(i,j,k)] = 0; //no longer part of ln2
				}
			}
		}
		
		if(layerD[index(i,j,k)] == 17){ //lp2
			if(checkMaskNeighbours2(i, j, k, 1, labelD) == false){
				labelD[index(i,j,k)] = 3;
				phiD[index(i,j,k)] = 3;
				layerD[index(i,j,k)] = 0; //no longer part of lp2
			}
			else{
				M = minMax(i, j, k,-1, 1, phiD, labelD);
				phiD[index(i,j,k)] = M+1;
				if(phiD[index(i,j,k)] < 1.5){
					layerD[index(i,j,k)] = 26; //add to sp1
				}
				else if(phiD[index(i,j,k)] >= 2.5){
					labelD[index(i,j,k)] = 3;
					phiD[index(i,j,k)] = 3;
					layerD[index(i,j,k)] = 0; //no longer part of lp2
				}
			}
		}
	}
}

__global__ void updateLevelSets1(float *phiD, int *layerD, int *labelD){
	int i = threadIdx.x + blockDim.x * blockIdx.x;
	int j = threadIdx.y + blockDim.y * blockIdx.y;
	int k = threadIdx.z + blockDim.z * blockIdx.z;
	if(i !=0 && j !=0 && i < HEIGHT-1 && j < WIDTH-1){
		if(layerD[index(i,j,k)] == 25){ //sz
			labelD[index(i,j,k)] = 0;
			layerD[index(i,j,k)] = 15; //add to lz
		}
		if(layerD[index(i,j,k)] == 24){ //sn1
			labelD[index(i,j,k)] = -1;
			layerD[index(i,j,k)] = 14; //add to ln1
			if(phiD[index(i+1,j,k)] == -3){
				phiD[index(i+1,j,k)] = phiD[index(i,j,k)] - 1;
				layerD[index(i+1,j,k)] = 23; //add to sn2
			}
			if(phiD[index(i,j+1,k)] == -3){
				phiD[index(i,j+1,k)] = phiD[index(i,j,k)] - 1;
				layerD[index(i,j+1,k)] = 23; //add to sn2
			}
			if(phiD[index(i,j,k+1)] == -3){
				phiD[index(i,j,k+1)] = phiD[index(i,j,k)] - 1;
				layerD[index(i,j,k+1)] = 23; //add to sn2
			}
			if(phiD[index(i-1,j,k)] == -3){
				phiD[index(i-1,j,k)] = phiD[index(i,j,k)] - 1;
				layerD[index(i-1,j,k)] = 23; //add to sn2
			}
			if(phiD[index(i,j-1,k)] == -3){
				phiD[index(i,j-1,k)] = phiD[index(i,j,k)] - 1;
				layerD[index(i,j-1,k)] = 23; //add to sn2
			}
			if(phiD[index(i,j,k-1)] == -3){
				phiD[index(i,j,k-1)] = phiD[index(i,j,k)] - 1;
				layerD[index(i,j,k-1)] = 23; //add to sn2
			}
		}
		if(layerD[index(i,j,k)] == 26){ //sp1
			labelD[index(i,j,k)] = 1;
			layerD[index(i,j,k)] = 16; ////add to lp1
			if(phiD[index(i+1,j,k)] == 3){
				phiD[index(i+1,j,k)] = phiD[index(i,j,k)] + 1;
				layerD[index(i+1,j,k)] = 27; //add to sp2
			}
			if(phiD[index(i,j+1,k)] == 3){
				phiD[index(i,j+1,k)] = phiD[index(i,j,k)] + 1;
				layerD[index(i,j+1,k)] = 27; //add to sp2
			}
			if(phiD[index(i,j,k+1)] == 3){
				phiD[index(i,j,k+1)] = phiD[index(i,j,k)] + 1;
				layerD[index(i,j,k+1)] = 27; //add to sp2
			}
			if(phiD[index(i-1,j,k)] == 3){
				phiD[index(i-1,j,k)] = phiD[index(i,j,k)] + 1;
				layerD[index(i-1,j,k)] = 27; //add to sp2
			}
			if(phiD[index(i,j-1,k)] == 3){
				phiD[index(i,j-1,k)] = phiD[index(i,j,k)] + 1;
				layerD[index(i,j-1,k)] = 27; //add to sp2
			}
			if(phiD[index(i,j,k-1)] == 3){
				phiD[index(i,j,k-1)] = phiD[index(i,j,k)] + 1;
				layerD[index(i,j,k-1)] = 27; //add to sn2
			}
		}
	}
}
	
__global__ void updateLevelSets2(int *layerD, int *labelD){	
	int i = threadIdx.x + blockDim.x * blockIdx.x;
	int j = threadIdx.y + blockDim.y * blockIdx.y;
	int k = threadIdx.z + blockDim.z * blockIdx.z;
	//no need to check if i and j are within range here
	if(layerD[index(i,j,k)] == 23){ //sn2
		labelD[index(i,j,k)] = -2;
		layerD[index(i,j,k)] = 13;  //add to ln2
	}
	else if(layerD[index(i,j,k)] == 27){ //sp2
		labelD[index(i,j,k)] = 2;
		layerD[index(i,j,k)] = 17; //add to lp2
	}
}