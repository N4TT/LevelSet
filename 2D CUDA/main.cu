#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include "main.h"
#include "update.h" //levelset process happens here
#include "EasyBMP.h" //library for reading bmp files
#include "IO.h" //handles input and stores output
#include <cstdio> //to calculate runtime
#include <ctime>  //to calculate runtime
using namespace std;

float image[HEIGHT][WIDTH] = { 0 }; //input -> image to be segmented
float phi[HEIGHT][WIDTH] = { 0 };
int init[HEIGHT][WIDTH] = { 0 };
int label[HEIGHT][WIDTH] = { 0 };
int zeroLevelSet[HEIGHT][WIDTH] = { 0 }; //output
int layer[HEIGHT][WIDTH]; //-> see main.h for details

int iterations;
float threshold, alpha, epsilon;

//to calculate runtime
clock_t start;
double duration;

//device arrays
float *phiD;
int *labelD;
int *layerD;
float *imageD;

__device__ float thresholdD, alphaD, epsilonD;

//fills init with circular seed point, returns 1 if success
int fillSphere(int seedX, int seedY, int radius){
	if(seedX < 0 || seedX > HEIGHT || seedY < 0 || seedY > WIDTH){
		printf("Wrong input to create a circular seed\n");
		printf("Coordinates out of range\n");
		return 0;
	}
	else if(radius < 1 || radius > HEIGHT/2 || radius > WIDTH/2){
		printf("Wrong input to create a circular seed\n");
		printf("Radius must be a positive integer less than min(width, height)/2\n");
		return 0;
	}
	for(int i = seedX - radius; i < seedX + radius; i++){
		for(int j = seedY - radius; j < seedY + radius; j++){
			if(sqrt((float)((seedX-i)*(seedX-i)+(seedY-j)*(seedY-j))) < radius){
				init[i][j] = 1;
			}
		}
	}
	return 1;
}

/* returns true if any neighbour of coordinates (i,j) in either
   init[][] (id = 1) or label[][] (id = 2) equals res */
bool checkMaskNeighbours(int i, int j, int res){
	if(init[i+1][j] == res)
		return true;
	else if(init[i-1][j] == res)
		return true;
	else if(init[i][j+1] == res)
		return true;
	else if(init[i][j-1] == res)
		return true;
	return false;
}

//add pixels to lists according to their label
void assignLabel(int i, int j, int level){
	switch(level){
	case 1:
		layer[i][j] = 16; //add to lp1
		label[i][j] = level;
		phi[i][j] = level;
		break;
	case 2:
		layer[i][j] = 17; //add to lp2
		label[i][j] = level;
		phi[i][j] = level;
		break;
	case -1:
		layer[i][j] = 14; //add to ln1
		label[i][j] = level;
		phi[i][j] = level;
		break;
	case -2:
		layer[i][j] = 13; //add to ln2
		label[i][j] = level;
		phi[i][j] = level;	
		break;
	}
}

void setLevels(int i, int j, int level){
	if(label[i+1][j] == 3){
		assignLabel(i+1, j, level);
	}
	if(label[i][j+1] == 3){
		assignLabel(i, j+1, level);
	}
	if(label[i-1][j] == 3){
		assignLabel(i-1, j, level);
	}
	if(label[i][j-1] == 3){
		assignLabel(i, j-1, level);
	}
	
	if(label[i+1][j] == -3){
		assignLabel(i+1, j, -level);
	}
	if(label[i][j+1] == -3){
		assignLabel(i, j+1, -level);
	}
	if(label[i-1][j] == -3){
		assignLabel(i-1, j, -level);
	}
	if(label[i][j-1] == -3){
		assignLabel(i, j-1, -level);
	}
}	

//initializes Ln2, Ln1, Lz, Lp1, Lp2 based on seed point(s)
void initialization(){
	for (int i = 0; i<HEIGHT; i++){
		for (int j = 0; j<WIDTH; j++){
			if(init[i][j] == 0){
				label[i][j] = 3; 
				phi[i][j] = 3;
			}
			else{
				label[i][j] = -3; 
				phi[i][j] = -3;
			}
		}
	}
	for (int i = 0; i<HEIGHT; i++){
		for (int j = 0; j<WIDTH; j++){
			if(init[i][j] == 1 && checkMaskNeighbours(i, j, 0) == true){
				layer[i][j] = 15; //add to lz
				label[i][j] = 0;
				phi[i][j] = 0;
			}
		}
	}
	for (int i = 0; i<HEIGHT; i++){
		for (int j = 0; j<WIDTH; j++){
			if(layer[i][j] == 15){ //lz
				setLevels(i, j, 1);
			}
		}
	}
	for (int i = 0; i<HEIGHT; i++){
		for (int j = 0; j<WIDTH; j++){
			if(layer[i][j] == 16){// lp1
				setLevels(i, j, 2);
			}
		}
	}
	for (int i = 0; i<HEIGHT; i++){
		for (int j = 0; j<WIDTH; j++){
			if(layer[i][j] == 14){ //ln1
				setLevels(i, j, 2);
			}
		}
	}
}

//allocate and copy data to device
void setUpDeviceArrays(){
	int err;
	const size_t arrSize = size_t(HEIGHT*WIDTH);
	err = hipMalloc((void**)&phiD, sizeof(float)*arrSize);
	if(err != hipSuccess){
		printf("phiD hipMalloc error: %d\n", err);
	}
	err = hipMalloc((void**)&labelD, sizeof(int)*arrSize);
	if(err != hipSuccess){
		printf("labelD hipMalloc error: %d\n", err);
	}
	err = hipMalloc((void**)&layerD, sizeof(int)*arrSize);
	if(err != hipSuccess){
		printf("layerD hipMalloc error: %d\n", err);
	}
	err = hipMalloc((void**)&imageD, sizeof(float)*arrSize);
	if(err != hipSuccess){
		printf("imageD hipMalloc error: %d\n", err);
	}
	
	err = hipMemcpy(phiD, phi, sizeof(float)*arrSize, hipMemcpyHostToDevice);
	if(err != hipSuccess){
		printf("phiD hipMemcpy error: %d\n", err);
	}
	err = hipMemcpy(labelD, label, sizeof(int)*arrSize, hipMemcpyHostToDevice);
	if(err != hipSuccess){
		printf("labelD hipMemcpy error: %d\n", err);
	}
	err = hipMemcpy(layerD, layer, sizeof(int)*arrSize, hipMemcpyHostToDevice);
	if(err != hipSuccess){
		printf("layerD hipMemcpy error: %d\n", err);
	}
	err = hipMemcpy(imageD, image, sizeof(float)*arrSize, hipMemcpyHostToDevice);
	if(err != hipSuccess){
		printf("imageD hipMemcpy error: %d\n", err);
	}
}

int main(int argc, char *argv[]){	printf("1");
	if(!getAndVerifyInput(argc, argv)){
		system("pause");
		return 0;
	}

	//read file
	BMP img;
	img.ReadFromFile("q1.bmp");
	readFile(img);
	
	if(fillSphere(250, 255, 10) == 0){
		system("pause");
		return 0;
	}
	
	initialization();
	setUpDeviceArrays(); //copy over data to device
	setVariablesInDevice<<<1,1>>>(threshold, epsilon, alpha, image);
	
	const dim3 BlockDim(16,16);
    dim3 GridDim;
    GridDim.x = (WIDTH + BlockDim.x - 1) / BlockDim.x;
    GridDim.y = (HEIGHT + BlockDim.y - 1) / BlockDim.y;
	
	printf("starting main loop\n");
	start = std::clock();
	for(int i=0; i<iterations+1; i++){
		if(i%100 == 0){
			printf("iteration: %i\n", i);
		}
		prepareUpdates1<<<GridDim, BlockDim>>>(phiD, layerD, imageD);
		prepareUpdates2<<<GridDim, BlockDim>>>(phiD, layerD, labelD);
		prepareUpdates3<<<GridDim, BlockDim>>>(phiD, layerD, labelD);
		prepareUpdates4<<<GridDim, BlockDim>>>(phiD, layerD, labelD);
		updateLevelSets1<<<GridDim, BlockDim>>>(phiD, layerD, labelD);
		updateLevelSets2<<<GridDim, BlockDim>>>(layerD, labelD);
	}
	duration = ( std::clock() - start ) / (double) CLOCKS_PER_SEC;
	printf("\nmain loop finished\n");
	printf("\ntime used: %f\n", duration);
	
	int err = hipMemcpy(label, labelD, sizeof(int)*(HEIGHT)*(WIDTH), hipMemcpyDeviceToHost);
	if(err != hipSuccess){
		printf("hipMemcpy error when writing to zeroLevelset: %d\n", err);
	}
	for (int i = 1; i<HEIGHT; i++){
		for (int j = 1; j<WIDTH; j++){
			if(label[i][j] == 0){ //lz
				zeroLevelSet[i][j] = 255;
			}
		}
	}
	
	writeFile(img, 1); //store label as image
	writeFile(img, 2); //store zerolevel set as image

	system("pause");
}
