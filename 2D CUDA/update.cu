#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include <cmath>
#include <hip/hip_runtime.h>
#include <exception>
#include <algorithm>
#include "main.h"
#include "update.h"
using namespace std;

extern float *phiD;
extern int *labelD;
extern int *layerD;
extern float *imageD;
extern __device__ float thresholdD, epsilonD, alphaD;

void __global__ setVariablesInDevice(float threshold, float epsilon, float alpha, float image[HEIGHT][WIDTH]){
	thresholdD = threshold;
	epsilonD = epsilon;
	alphaD = alpha;
}

//nvcc --machine 32 -arch sm_20 main.cu update.cu IO.cu EasyBMP.cpp 

//Returns either max or min (based on greaterOrLess) of the neighbours, with values less or greater than checkAgainst
__device__ float follow(int i, int j, int greaterOrLess, int checkAgainst, float *phiD, int *labelD){
	float fResult = checkAgainst;
	if(greaterOrLess == 1){
		if(labelD[(i+1)*WIDTH+j] >= fResult){
			fResult = phiD[(i+1)*WIDTH+j];
		}
		if(labelD[i*WIDTH+(j+1)] >= fResult){
			fResult = phiD[i*WIDTH + (j+1)];
		}
		if(labelD[(i-1)*WIDTH + j] >= fResult){
			fResult = phiD[(i-1)*WIDTH+j];
		}
		if(labelD[i*WIDTH+(j-1)] >= fResult){
			fResult = phiD[i*WIDTH+(j-1)];
		}
	}
	else if(greaterOrLess == -1){
		if(labelD[(i+1)*WIDTH+j] <= fResult){
			fResult = phiD[(i+1)*WIDTH+j];
		}
		if(labelD[i*WIDTH+(j+1)] <= fResult){
			fResult = phiD[i*WIDTH+(j+1)];
		}
		if(labelD[(i-1)*WIDTH+j] <= fResult){
			fResult = phiD[(i-1)*WIDTH+j];
		}
		if(labelD[i*WIDTH+(j-1)] <= fResult){
			fResult= phiD[i*WIDTH+(j-1)];
		}
	}
	return fResult;	
}

__device__ bool checkMaskNeighbours2(int i, int j, short res, int *labelD){
	if(labelD[(i+1)*WIDTH+j] == res)
		return true;
	else if(labelD[(i-1)*WIDTH+j] == res)
		return true;
	else if(labelD[i*WIDTH+(j+1)] == res)
		return true;
	else if(labelD[i*WIDTH+(j-1)] == res)
		return true;	
	return false;
}

__device__ float speedFunction(int i, int j, float *phiD, float *imageD){ 
	//calculate data term
	float data = epsilonD - abs(imageD[i*WIDTH+j] - thresholdD); //the data term (based on pixel intensity)
	//calculate first order derivatives
	float dx = (phiD[(i+1)*WIDTH+j] - phiD[(i-1)*WIDTH+ j]) / 2;
	float dy = (phiD[i*WIDTH+(j+1)] - phiD[i*WIDTH+(j-1)]) / 2;
	float dxPlus = phiD[(i+1)*WIDTH+j] - phiD[i*WIDTH+j];
	float dyPlus = phiD[i*WIDTH+(j+1)] - phiD[i*WIDTH+j];
	float dxMinus = phiD[i*WIDTH+j] - phiD[(i-1)*WIDTH+ j];
	float dyMinus = phiD[i*WIDTH+j] - phiD[i*WIDTH+(j-1)];
	//calculate second order derivatives
	float dxPlusY = (phiD[(i+1)*WIDTH+(j+1)] - phiD[(i-1)*WIDTH+(j+1)])/2;
	float dxMinusY = (phiD[(i+1)*WIDTH+(j-1)] - phiD[(i-1)*WIDTH+(j-1)])/2;
	float dyPlusX = (phiD[(i+1)*WIDTH+(j+1)] - phiD[(i+1)*WIDTH+(j-1)])/2;
	float dyMinusX = (phiD[(i-1)*WIDTH+(j+1)] - phiD[(i-1)*WIDTH+(j-1)])/2;
	//calculate normals
	float nPlusX = dxPlus / sqrt(dxPlus*dxPlus + pow((dyPlusX + dy) / 2, 2));
	float nPlusY = dyPlus / sqrt(dyPlus*dyPlus + pow((dxPlusY + dx) / 2, 2));
	float nMinusX = dxMinus / sqrt(dxMinus * dxMinus + pow((dyMinusX + dy) / 2, 2));
	float nMinusY = dyMinus / sqrt(dyMinus * dyMinus + pow((dxMinusY + dx) / 2, 2));
	//calculate curvature
	float curvature = (nPlusX - nMinusX) + (nPlusY - nMinusY);
	//calculate the speeed
	float speed = -alphaD*data + (1.0f-alphaD)*(curvature/4.0f); //divided by 4 to narmalize (max(curvature) = 4)
	//clamp speed
	if(speed > 1.0f){
		speed = 1.0f;
	}
	if(speed < -1.0f){
		speed = -1.0f;
	}
	return speed;
}

__global__ void prepareUpdates1(float *phiD, int *layerD, float *imageD){
	int i = threadIdx.x + blockDim.x * blockIdx.x;
	int j = threadIdx.y + blockDim.y * blockIdx.y;
	if(layerD[i*WIDTH+j] == 15){ //lz
		phiD[i*WIDTH+j] += speedFunction(i,j, phiD, imageD);
		if(phiD[i*WIDTH+j] >= 0.5){
			layerD[i*WIDTH+j] = 26; //add to sp1
			
		}
		else if(phiD[i*WIDTH+j] < -0.5){
			layerD[i*WIDTH+j] = 24; //add to sn1
		}
	}
}

__global__ void prepareUpdates2(float *phiD, int *layerD, int *labelD){
	int i = threadIdx.x + blockDim.x * blockIdx.x;
	int j = threadIdx.y + blockDim.y * blockIdx.y;
	if(i !=0 && j !=0 && i < HEIGHT-1 && j < WIDTH-1){
		float M = 0;
		if(layerD[i*WIDTH+j] == 14){ //ln1
			if(checkMaskNeighbours2(i, j, 0, labelD) == false){
				layerD[i*WIDTH+j] = 23; //add to sn2
			}
			else{
				M = follow(i, j, 1, 0, phiD, labelD);
				phiD[i*WIDTH+j] = M-1;
				if(phiD[i*WIDTH+j] >= -0.5){
					layerD[i*WIDTH+j] = 25; //add to sz
				}
				else if(phiD[i*WIDTH+j] < -1.5){
					layerD[i*WIDTH+j] = 23; //add to sn2
				}
			}
		}
	}
}

__global__ void prepareUpdates3(float *phiD, int *layerD, int *labelD){
	int i = threadIdx.x + blockDim.x * blockIdx.x;
	int j = threadIdx.y + blockDim.y * blockIdx.y;
	if(i !=0 && j !=0 && i < HEIGHT-1 && j < WIDTH-1){
		float M = 0;
		if(layerD[i*WIDTH+j] == 16){ //lp1
			if(checkMaskNeighbours2(i, j, 0, labelD) == false){
				layerD[i*WIDTH+j] = 27; //add to sp2
			}
			else{
				M = follow(i, j, -1, 0, phiD, labelD);
				phiD[i*WIDTH+j] = M+1;
				if(phiD[i*WIDTH+j] < 0.5){
					layerD[i*WIDTH+j] = 25; //add to sz
				}
				else if(phiD[i*WIDTH+j] >= 1.5){
					layerD[i*WIDTH+j] = 27; //add to sp2
				}
			}
		}
	}
}

__global__ void prepareUpdates4(float *phiD, int *layerD, int *labelD){
	int i = threadIdx.x + blockDim.x * blockIdx.x;
	int j = threadIdx.y + blockDim.y * blockIdx.y;
	if(i !=0 && j !=0 && i < HEIGHT-1 && j < WIDTH-1){
		float M = 0;
		if(layerD[i*WIDTH+j] == 13){ //ln2
			if(checkMaskNeighbours2(i, j, -1, labelD) == false){
				labelD[i*WIDTH+j] = -3;
				phiD[i*WIDTH+j] = -3;
				layerD[i*WIDTH+j] = 0; //no longer part of ln2
			}
			else{
				M = follow(i, j, 1, -1, phiD, labelD);
				phiD[i*WIDTH+j] = M-1;
				if(phiD[i*WIDTH+j] >= -1.5){
					layerD[i*WIDTH+j] = 24; //add to sn1
				}
				else if(phiD[i*WIDTH+j] < -2.5){
					labelD[i*WIDTH+j] = -3;
					phiD[i*WIDTH+j] = -3;
					layerD[i*WIDTH+j] = 0; //no longer part of ln2
				}
			}
		}
		
		if(layerD[i*WIDTH+j] == 17){ //lp2
			if(checkMaskNeighbours2(i, j, 1, labelD) == false){
				labelD[i*WIDTH+j] = 3;
				phiD[i*WIDTH+j] = 3;
				layerD[i*WIDTH+j] = 0; //no longer part of lp2
			}
			else{
				M = follow(i, j, -1, 1, phiD, labelD);
				phiD[i*WIDTH+j] = M+1;
				if(phiD[i*WIDTH+j] < 1.5){
					layerD[i*WIDTH+j] = 26; //add to sp1
				}
				else if(phiD[i*WIDTH+j] >= 2.5){
					labelD[i*WIDTH+j] = 3;
					phiD[i*WIDTH+j] = 3;
					layerD[i*WIDTH+j] = 0; //no longer part of lp2
				}
			}
		}
	}
}

__global__ void updateLevelSets1(float *phiD, int *layerD, int *labelD){
	int i = threadIdx.x + blockDim.x * blockIdx.x;
	int j = threadIdx.y + blockDim.y * blockIdx.y;
	if(i !=0 && j !=0 && i < HEIGHT-1 && j < WIDTH-1){
		if(layerD[i*WIDTH+j] == 25){ //sz
			labelD[i*WIDTH+j] = 0;
			layerD[i*WIDTH+j] = 15; //add to lz
		}
		if(layerD[i*WIDTH+j] == 24){ //sn1
			labelD[i*WIDTH+j] = -1;
			layerD[i*WIDTH+j] = 14; //add to ln1
			if(phiD[(i+1)*WIDTH+ j] == -3){
				phiD[(i+1)*WIDTH+ j] = phiD[i*WIDTH+j] - 1;
				layerD[(i+1)*WIDTH+ j] = 23; //add to sn2
			}
			if(phiD[i*WIDTH+ (j+1)] == -3){
				phiD[i*WIDTH+ (j+1)] = phiD[i*WIDTH+j] - 1;
				layerD[i*WIDTH+ (j+1)] = 23; //add to sn2
			}
			if(phiD[(i-1)*WIDTH+ j] == -3){
				phiD[(i-1)*WIDTH+ j] = phiD[i*WIDTH+j] - 1;
				layerD[(i-1)*WIDTH+ j] = 23; //add to sn2
			}
			if(phiD[i*WIDTH+ (j-1)] == -3){
				phiD[i*WIDTH+ (j-1)] = phiD[i*WIDTH+j] - 1;
				layerD[i*WIDTH+ (j-1)] = 23; //add to sn2
			}
		}
		if(layerD[i*WIDTH+j] == 26){ //sp1
			labelD[i*WIDTH+j] = 1;
			layerD[i*WIDTH+j] = 16; ////add to lp1
			if(phiD[(i+1)*WIDTH+ j] == 3){
				phiD[(i+1)*WIDTH+ j] = phiD[i*WIDTH+j] + 1;
				layerD[(i+1)*WIDTH+ j] = 27; //add to sp2
			}
			if(phiD[i*WIDTH+ (j+1)] == 3){
				phiD[i*WIDTH+ (j+1)] = phiD[i*WIDTH+j] + 1;
				layerD[i*WIDTH+ (j+1)] = 27; //add to sp2
			}
			if(phiD[(i-1)*WIDTH+ j] == 3){
				phiD[(i-1)*WIDTH+ j] = phiD[i*WIDTH+j] + 1;
				layerD[(i-1)*WIDTH+ j] = 27; //add to sp2
			}
			if(phiD[i*WIDTH+ (j-1)] == 3){
				phiD[i*WIDTH+ (j-1)] = phiD[i*WIDTH+j] + 1;
				layerD[i*WIDTH+ (j-1)] = 27; //add to sp2
			}
		}
	}
}
	
__global__ void updateLevelSets2(int *layerD, int *labelD){	
	int i = threadIdx.x + blockDim.x * blockIdx.x;
	int j = threadIdx.y + blockDim.y * blockIdx.y;
	//no need to check if i and j are within range here
	if(layerD[i*WIDTH+j] == 23){ //sn2
		labelD[i*WIDTH+j] = -2;
		layerD[i*WIDTH+j] = 13;  //add to ln2
	}
	if(layerD[i*WIDTH+j] == 27){ //sp2
		labelD[i*WIDTH+j] = 2;
		layerD[i*WIDTH+j] = 17; //add to lp2
	}
}